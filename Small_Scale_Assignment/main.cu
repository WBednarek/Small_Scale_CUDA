
#include "hip/hip_runtime.h"
#include ""
#include <iostream>
#include <stdio.h>
#include <cstdlib>
#include <vector>
#include <string>
#include "ReadMatrixCSR.h"
#include "ReadMatrixELL.h"
#include "SimulationAndTheTests.h"
#include "SimulationAndTheTests.cpp"

#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <cstdlib>
#include <hip/hip_runtime.h>
#include <helper_cuda_drvapi.h>
#include <drvapi_error_string.h>
#include  "deviceQueryDrv.h"



/**
Zmienc nazwu

*/



void displayValues(std::vector<int> JA, std::vector<int> IRP, std::vector<double> AS )
{

	//Output file in to the project folder
	std::ofstream storeArray("output4.txt");
	if (storeArray.is_open())
	{
		storeArray << "JA has folllowing values: ";
		for (std::vector<int>::const_iterator i = JA.begin(); i != JA.end(); ++i)
		{
			storeArray << *i << ' ';
		}

		storeArray << std::endl;

		

		storeArray << "IRP has folllowing values: ";
		for (std::vector<int>::const_iterator i = IRP.begin(); i != IRP.end(); ++i)
		{
			storeArray << *i << ' ';
		}

		storeArray << std::endl;

		storeArray << "AS has folllowing values: ";
		for (std::vector<double>::const_iterator i = AS.begin(); i != AS.end(); ++i)
		{
			storeArray << *i << ' ';
		}

		storeArray << std::endl;

	}

}



void displayOneDimensionalELLValues(std::vector<int> JA, std::vector<double> AS)
{

	//Output file in to the project folder
	std::ofstream storeArray("ELLOneDimesionalNEW.txt");
	if (storeArray.is_open())
	{
		storeArray << "JA has folllowing values: ";
		for (std::vector<int>::const_iterator i = JA.begin(); i != JA.end(); ++i)
		{
			storeArray << *i << ' ';
		}

		storeArray << std::endl;


		storeArray << "AS has the following values: ";
		for (std::vector<double>::const_iterator i = AS.begin(); i != AS.end(); ++i)
		{
			storeArray << *i << ' ';
		}

		storeArray << std::endl;

	}

}

void readCudaParameters()
{
	hipDevice_t dev;
	int deviceCount = 0;

	// note your project will need to link with cuda.lib files on windows
	printf("CUDA Device Query (Driver API) statically linked version \n");

	hipError_t error_id = hipInit(0);

	if (error_id != hipSuccess)
	{
		printf("hipInit(0) returned %d\n-> %s\n", error_id, getCudaDrvErrorString(error_id));
		printf("Result = FAIL\n");
		exit(EXIT_FAILURE);
	}

	error_id = hipGetDeviceCount(&deviceCount);

	if (error_id != hipSuccess)
	{
		printf("hipGetDeviceCount returned %d\n-> %s\n", (int)error_id, getCudaDrvErrorString(error_id));
		printf("Result = FAIL\n");
		exit(EXIT_FAILURE);
	}

	// This function call returns 0 if there are no CUDA capable devices.
	if (deviceCount == 0)
	{
		printf("There are no available device(s) that support CUDA\n");
	}
	else
	{
		printf("Detected %d CUDA Capable device(s)\n", deviceCount);
	}


	for (dev = 0; dev < deviceCount; ++dev)
	{
		int warpSize;
		getCudaAttribute<int>(&warpSize, hipDeviceAttributeWarpSize, dev);
		std::cout << "Warp size is: " << warpSize << std::endl;
	}

}





int main(int argc, char *argv[])
{
    
	std::vector<std::string> matriresList;
	//char deviceName[256];
	std::cout << "Running CUDA simulation" << std::endl;
	std::cout << "List of matrices running on:" << std::endl;
	std::string currentMattix = argv[1];
	std::cout << "Argument 1 is: " << currentMattix << std::endl;

	readCudaParameters();



	//displayValues(matrixCSR.getJA(), matrixCSR.getIRP(), matrixELL.getAS());

	//displayOneDimensionalELLValues(matrixELL.getJA(), matrixELL.getAS());
	//displayValues(matrixCSR.getJA(), matrixCSR.getIRP() , matrixCSR.getAS());

	/**
	//Start Parallel computation
	*/

	

	//sim.template runCUDA<ReadMatrixCSR>(matrixCSR, numberOfThreads, sizeOfBlock, maxNumberOfBlocks, timeToComplete);

	/*"west2021.mtx"

	"dc1.mtx",
	"af23560.mtx",
	mhd4800a.mtx",
	"amazon0302.mtx",
	"bcsstk17.mtx",
	"cop20k_A.mtx"
	"FEM_3D_thermal1.mtx",
	"lung2.mtx",
	"mac_econ_fwd500.mtx",
	"olafu.mtx",
	"raefsky2.mtx",
*/	
	/*
	done "adder_dcop_32.mtx", 
	"olafu_b.mtx",
	"olm1000.mtx",
	"mhda416.mtx",
	"mcfe.mtx",
	"rdist2.mtx",
	"cavity10.mtx",
	"cage4.mtx",
	*/
	std::vector<std::string> matricesNames = { 
		
		
	
		"roadNet-PA.mtx",
		"thermal1.mtx",

		"thermomech_TK.mtx",
		 };





	int simulationRuns = 10;
	unsigned int numberOfThreads = 4;
	unsigned int sizeOfBlock = 64;
	unsigned int maxNumberOfBlocks = 4096;

	for (auto it : matricesNames)
	{
		// Read input matrices
		ReadMatrixCSR matrixCSR(it);
		ReadMatrixELL matrixELL(it);
		SimulationAndTheTests<ReadMatrixCSR> simCSR;
		SimulationAndTheTests<ReadMatrixELL> simELLPack;
		//std::cout<<"ELL SIZE: "<<sizeof(matrixELL)<<std::endl;

		//OpenMP Run
		simCSR.runOpenMP(matrixCSR, numberOfThreads, simulationRuns);
		simELLPack.runOpenMP(matrixELL, numberOfThreads, simulationRuns);

		//CUDA Run
		simCSR.runCUDA(matrixCSR, numberOfThreads, sizeOfBlock, maxNumberOfBlocks, simulationRuns);
		simELLPack.runCUDA(matrixELL, numberOfThreads, sizeOfBlock, maxNumberOfBlocks, simulationRuns);

	}

	
	

	system("pause");
    return 0;
}

