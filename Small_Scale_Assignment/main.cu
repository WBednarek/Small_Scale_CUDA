
#include "hip/hip_runtime.h"
#include ""
#include <iostream>
#include <stdio.h>
#include <cstdlib>
#include <vector>
#include <string>
#include "ReadMatrixCSR.h"
#include "ReadMatrixELL.h"
#include "SimulationAndTheTests.h"
#include "SimulationAndTheTests.cpp"

#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <cstdlib>
#include <hip/hip_runtime.h>
#include <helper_cuda_drvapi.h>
#include <drvapi_error_string.h>
#include  "deviceQueryDrv.h"



/**
Zmienc nazwu

*/



void displayValues(std::vector<int> JA, std::vector<int> IRP, std::vector<double> AS )
{

	//Output file in to the project folder
	std::ofstream storeArray("output4.txt");
	if (storeArray.is_open())
	{
		storeArray << "JA has folllowing values: ";
		for (std::vector<int>::const_iterator i = JA.begin(); i != JA.end(); ++i)
		{
			storeArray << *i << ' ';
		}

		storeArray << std::endl;

		

		storeArray << "IRP has folllowing values: ";
		for (std::vector<int>::const_iterator i = IRP.begin(); i != IRP.end(); ++i)
		{
			storeArray << *i << ' ';
		}

		storeArray << std::endl;

		storeArray << "AS has folllowing values: ";
		for (std::vector<double>::const_iterator i = AS.begin(); i != AS.end(); ++i)
		{
			storeArray << *i << ' ';
		}

		storeArray << std::endl;

	}

}



void displayOneDimensionalELLValues(std::vector<int> JA, std::vector<double> AS)
{

	//Output file in to the project folder
	std::ofstream storeArray("ELLOneDimesionalNEW.txt");
	if (storeArray.is_open())
	{
		storeArray << "JA has folllowing values: ";
		for (std::vector<int>::const_iterator i = JA.begin(); i != JA.end(); ++i)
		{
			storeArray << *i << ' ';
		}

		storeArray << std::endl;


		storeArray << "AS has the following values: ";
		for (std::vector<double>::const_iterator i = AS.begin(); i != AS.end(); ++i)
		{
			storeArray << *i << ' ';
		}

		storeArray << std::endl;

	}

}

void readCudaParameters()
{
	hipDevice_t dev;
	int deviceCount = 0;

	// note your project will need to link with cuda.lib files on windows
	printf("CUDA Device Query (Driver API) statically linked version \n");

	hipError_t error_id = hipInit(0);

	if (error_id != hipSuccess)
	{
		printf("hipInit(0) returned %d\n-> %s\n", error_id, getCudaDrvErrorString(error_id));
		printf("Result = FAIL\n");
		exit(EXIT_FAILURE);
	}

	error_id = hipGetDeviceCount(&deviceCount);

	if (error_id != hipSuccess)
	{
		printf("hipGetDeviceCount returned %d\n-> %s\n", (int)error_id, getCudaDrvErrorString(error_id));
		printf("Result = FAIL\n");
		exit(EXIT_FAILURE);
	}

	// This function call returns 0 if there are no CUDA capable devices.
	if (deviceCount == 0)
	{
		printf("There are no available device(s) that support CUDA\n");
	}
	else
	{
		printf("Detected %d CUDA Capable device(s)\n", deviceCount);
	}


	for (dev = 0; dev < deviceCount; ++dev)
	{
		int warpSize;
		getCudaAttribute<int>(&warpSize, hipDeviceAttributeWarpSize, dev);
		std::cout << "Warp size is: " << warpSize << std::endl;
	}

}





int main(int argc, char *argv[])
{

	std::vector<std::string> matriresList;
	std::cout << "USAGE: put all matrices into your main project folder \nor mass one matrix as program parameter\n Program gives .xls files in the output in program main directory folder\n" << std::endl;
	
	std::string currentMattix = argv[1];
	std::cout << "Argument 1 is: " << currentMattix << std::endl;

	readCudaParameters();

	std::vector<std::string> matricesNames = { 	
		"cage4.mtx",
		
		 };


	int simulationRuns = 10;
	unsigned int numberOfThreads = 4;
	unsigned int sizeOfBlock = 64;
	unsigned int maxNumberOfBlocks = 4096;

	std::cout << "The number of simulations repetitions: " << simulationRuns << std::endl << std::endl;;
	/**
	//Start Parallel computation
	*/
	for (auto it : matricesNames)
	{
		// Read input matrices
		ReadMatrixCSR matrixCSR(it);
		ReadMatrixELL matrixELL(it);
		SimulationAndTheTests<ReadMatrixCSR> simCSR;
		SimulationAndTheTests<ReadMatrixELL> simELLPack;
		//std::cout<<"ELL SIZE: "<<sizeof(matrixELL)<<std::endl;

		//OpenMP Run
		simCSR.runOpenMP(matrixCSR, numberOfThreads, simulationRuns);
		simELLPack.runOpenMP(matrixELL, numberOfThreads, simulationRuns);

		//CUDA Run
		simCSR.runCUDA(matrixCSR, numberOfThreads, sizeOfBlock, maxNumberOfBlocks, simulationRuns);
		simELLPack.runCUDA(matrixELL, numberOfThreads, sizeOfBlock, maxNumberOfBlocks, simulationRuns);

	}

	system("pause");
    return 0;
}

