
#include "hip/hip_runtime.h"
#include ""
#include <iostream>
#include <stdio.h>
#include <cstdlib>
#include <vector>
#include <string>
#include "ReadMatrixCSR.h"
#include "ReadMatrixELL.h"
#include "SimulationAndTheTests.h"
#include "SimulationAndTheTests.cpp"

#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <cstdlib>
#include <hip/hip_runtime.h>
#include <helper_cuda_drvapi.h>
#include <drvapi_error_string.h>
#include  "deviceQueryDrv.h"



/**
Zmienc nazwu

*/



void displayValues(std::vector<int> JA, std::vector<int> IRP, std::vector<double> AS )
{

	//Output file in to the project folder
	std::ofstream data("output4.txt");
	if (data.is_open())
	{
		data << "JA has folllowing values: ";
		for (std::vector<int>::const_iterator i = JA.begin(); i != JA.end(); ++i)
		{
			data << *i << ' ';
		}

		data << std::endl;

		

		data << "IRP has folllowing values: ";
		for (std::vector<int>::const_iterator i = IRP.begin(); i != IRP.end(); ++i)
		{
			data << *i << ' ';
		}

		data << std::endl;

		data << "AS has folllowing values: ";
		for (std::vector<double>::const_iterator i = AS.begin(); i != AS.end(); ++i)
		{
			data << *i << ' ';
		}

		data << std::endl;

	}

}



void displayOneDimensionalELLValues(std::vector<int> JA, std::vector<double> AS)
{

	//Output file in to the project folder
	std::ofstream data("ELLOneDimesionalNEW.txt");
	if (data.is_open())
	{
		data << "JA has folllowing values: ";
		for (std::vector<int>::const_iterator i = JA.begin(); i != JA.end(); ++i)
		{
			data << *i << ' ';
		}

		data << std::endl;


		data << "AS has the following values: ";
		for (std::vector<double>::const_iterator i = AS.begin(); i != AS.end(); ++i)
		{
			data << *i << ' ';
		}

		data << std::endl;

	}

}

void readCudaParameters()
{
	hipDevice_t dev;
	int deviceCount = 0;

	// note your project will need to link with cuda.lib files on windows
	printf("CUDA Device Query (Driver API) statically linked version \n");

	hipError_t error_id = hipInit(0);

	if (error_id != hipSuccess)
	{
		printf("hipInit(0) returned %d\n-> %s\n", error_id, getCudaDrvErrorString(error_id));
		printf("Result = FAIL\n");
		exit(EXIT_FAILURE);
	}

	error_id = hipGetDeviceCount(&deviceCount);

	if (error_id != hipSuccess)
	{
		printf("hipGetDeviceCount returned %d\n-> %s\n", (int)error_id, getCudaDrvErrorString(error_id));
		printf("Result = FAIL\n");
		exit(EXIT_FAILURE);
	}

	// This function call returns 0 if there are no CUDA capable devices.
	if (deviceCount == 0)
	{
		printf("There are no available device(s) that support CUDA\n");
	}
	else
	{
		printf("Detected %d CUDA Capable device(s)\n", deviceCount);
	}


	for (dev = 0; dev < deviceCount; ++dev)
	{
		int warpSize;
		getCudaAttribute<int>(&warpSize, hipDeviceAttributeWarpSize, dev);
		std::cout << "Warp size is: " << warpSize << std::endl;
	}

}





int main(int argc, char *argv[])
{
    
	std::vector<std::string> matriresList;
	//char deviceName[256];
	std::cout << "Running CUDA simulation" << std::endl;
	std::cout << "List of matrices running on:" << std::endl;
	std::string currentMattix = argv[1];
	std::cout << "Argument 1 is: " << currentMattix << std::endl;

	readCudaParameters();

	// Read input matrices
	ReadMatrixCSR matrixCSR(currentMattix);
	ReadMatrixELL matrixELL(currentMattix);
	SimulationAndTheTests<ReadMatrixCSR> simCSR;
	SimulationAndTheTests<ReadMatrixELL> simELLPack;
	//std::cout<<"ELL SIZE: "<<sizeof(matrixELL)<<std::endl;

	//displayValues(matrixCSR.getJA(), matrixCSR.getIRP(), matrixELL.getAS());

	//displayOneDimensionalELLValues(matrixELL.getJA(), matrixELL.getAS());
	//displayValues(matrixCSR.getJA(), matrixCSR.getIRP() , matrixCSR.getAS());

	/**
	//Start Parallel computation
	*/

	

	//sim.template runCUDA<ReadMatrixCSR>(matrixCSR, numberOfThreads, sizeOfBlock, maxNumberOfBlocks, timeToComplete);


	

	int simulationRuns = 100;
	unsigned int numberOfThreads = 4;
	unsigned int sizeOfBlock = 64;
	unsigned int maxNumberOfBlocks = 4096;

	//OpenMP Run
	simCSR.runOpenMP(matrixCSR, numberOfThreads, simulationRuns);
	simELLPack.runOpenMP(matrixELL, numberOfThreads, simulationRuns);

	//CUDA Run
	//simCSR.runCUDA(matrixCSR, numberOfThreads, sizeOfBlock, maxNumberOfBlocks, simulationRuns);
	simELLPack.runCUDA(matrixELL, numberOfThreads, sizeOfBlock, maxNumberOfBlocks, simulationRuns);
	

	system("pause");
    return 0;
}

