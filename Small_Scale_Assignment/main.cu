
#include "hip/hip_runtime.h"
#include ""
#include <iostream>
#include <stdio.h>
#include <cstdlib>
#include <vector>
#include <string>
#include "ReadMatrixCSR.h"
#include "ReadMatrixELL.h"
#include "SimulationAndTheTests.h"
#include "SimulationAndTheTests.cpp"

#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <cstdlib>
#include <hip/hip_runtime.h>
#include <helper_cuda_drvapi.h>
#include <drvapi_error_string.h>
#include  "deviceQueryDrv.h"



/**
Zmienc nazwu

*/



void displayValues(std::vector<int> JA, std::vector<int> IRP, std::vector<double> AS )
{

	//Output file in to the project folder
	std::ofstream data("output4.txt");
	if (data.is_open())
	{
		data << "JA has folllowing values: ";
		for (std::vector<int>::const_iterator i = JA.begin(); i != JA.end(); ++i)
		{
			data << *i << ' ';
		}

		data << std::endl;

		

		data << "IRP has folllowing values: ";
		for (std::vector<int>::const_iterator i = IRP.begin(); i != IRP.end(); ++i)
		{
			data << *i << ' ';
		}

		data << std::endl;

		data << "AS has folllowing values: ";
		for (std::vector<double>::const_iterator i = AS.begin(); i != AS.end(); ++i)
		{
			data << *i << ' ';
		}

		data << std::endl;

	}

}



void displayOneDimensionalELLValues(std::vector<int> JA, std::vector<double> AS)
{

	//Output file in to the project folder
	std::ofstream data("ELLOneDimesionalNEW.txt");
	if (data.is_open())
	{
		data << "JA has folllowing values: ";
		for (std::vector<int>::const_iterator i = JA.begin(); i != JA.end(); ++i)
		{
			data << *i << ' ';
		}

		data << std::endl;


		data << "AS has the following values: ";
		for (std::vector<double>::const_iterator i = AS.begin(); i != AS.end(); ++i)
		{
			data << *i << ' ';
		}

		data << std::endl;

	}

}

void readCudaParameters()
{
	hipDevice_t dev;
	int deviceCount = 0;

	// note your project will need to link with cuda.lib files on windows
	printf("CUDA Device Query (Driver API) statically linked version \n");

	hipError_t error_id = hipInit(0);

	if (error_id != hipSuccess)
	{
		printf("hipInit(0) returned %d\n-> %s\n", error_id, getCudaDrvErrorString(error_id));
		printf("Result = FAIL\n");
		exit(EXIT_FAILURE);
	}

	error_id = hipGetDeviceCount(&deviceCount);

	if (error_id != hipSuccess)
	{
		printf("hipGetDeviceCount returned %d\n-> %s\n", (int)error_id, getCudaDrvErrorString(error_id));
		printf("Result = FAIL\n");
		exit(EXIT_FAILURE);
	}

	// This function call returns 0 if there are no CUDA capable devices.
	if (deviceCount == 0)
	{
		printf("There are no available device(s) that support CUDA\n");
	}
	else
	{
		printf("Detected %d CUDA Capable device(s)\n", deviceCount);
	}


	for (dev = 0; dev < deviceCount; ++dev)
	{
		int warpSize;
		getCudaAttribute<int>(&warpSize, hipDeviceAttributeWarpSize, dev);
		std::cout << "Warp size is: " << warpSize << std::endl;
	}

}





int main(int argc, char *argv[])
{
    
	std::vector<std::string> matriresList;
	//char deviceName[256];
	std::cout << "Running CUDA simulation" << std::endl;
	std::cout << "List of matrices running on:" << std::endl;
	std::string currentMattix = argv[1];
	std::cout << "Argument 1 is: " << currentMattix << std::endl;

	readCudaParameters();

	// Read input matrices
	ReadMatrixCSR matrixCSR(currentMattix);
	ReadMatrixELL matrixELL(currentMattix);
	SimulationAndTheTests<ReadMatrixCSR> simCSR;
	SimulationAndTheTests<ReadMatrixELL> simELLPack;
	//std::cout<<"ELL SIZE: "<<sizeof(matrixELL)<<std::endl;

	//displayValues(matrixCSR.getJA(), matrixCSR.getIRP(), matrixELL.getAS());

	//displayOneDimensionalELLValues(matrixELL.getJA(), matrixELL.getAS());
	//displayValues(matrixCSR.getJA(), matrixCSR.getIRP() , matrixCSR.getAS());

	/**
	//Start Parallel computation
	*/

	

	//sim.template runCUDA<ReadMatrixCSR>(matrixCSR, numberOfThreads, sizeOfBlock, maxNumberOfBlocks, timeToComplete);


	std::vector<std::string> matricesNames = { "adder_dcop_32.mtx",
		"af23560.mtx",
		"af_1_k101_b.mtx",
		"amazon0302.mtx",
		"bcsstk17.mtx",
		"cage4.mtx",
		 "cavity10.mtx",
		"cavity10_b.mtx",
		"cavity10_x.mtx",
		"cop20k_A.mtx",
		"dc1.mtx",
		"dc1_b.mtx",
		"FEM_3D_thermal1.mt",
		"lung2.mtx",
		"mac_econ_fwd500.mt",
		"mcfe.mtx",
		"mhd4800a.mtx",
		"mhda416.mtx",
		"olafu.mtx",
		"olafu_b.mtx",
		"olm1000.mtx",
		"PR02R_b.mtx",
		"PR02R_x.mtx",
		"raefsky2.mtx",
		"raefsky2_b.mtx",
		"rdist2.mtx",
		"roadNet - PA.mtx",
		"thermal1.mtx",
		"thermal1_b.mtx",
		"thermal2_b.mtx",
		"thermomech_TK.mtx",
		"west2021.mtx" };

	int simulationRuns = 100;
	unsigned int numberOfThreads = 4;
	unsigned int sizeOfBlock = 64;
	unsigned int maxNumberOfBlocks = 4096;

	//OpenMP Run
	simCSR.runOpenMP(matrixCSR, numberOfThreads, simulationRuns);
	simELLPack.runOpenMP(matrixELL, numberOfThreads, simulationRuns);

	//CUDA Run
	//simCSR.runCUDA(matrixCSR, numberOfThreads, sizeOfBlock, maxNumberOfBlocks, simulationRuns);
	simELLPack.runCUDA(matrixELL, numberOfThreads, sizeOfBlock, maxNumberOfBlocks, simulationRuns);
	

	system("pause");
    return 0;
}

