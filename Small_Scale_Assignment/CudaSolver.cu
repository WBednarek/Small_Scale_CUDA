#include "hip/hip_runtime.h"
#include "CudaSolver.h"


/**
*
* @param M Numner of rows
* @param JA Array of column indieces
* @param IRP Array of pointers to row start
* @param AS Array of cooefficients
* @param OUT Solution vector
* @param blockAmount
*/


__global__ void CSRCudaMatrixVectorProduct(const int &M, const int * JA, const int * IRP, const double * AS, double * IN, double * OUT)
{

	//Based on local memory
	extern __shared__ double data[];

	unsigned int warpSize = 32;

	unsigned int threadIndex = threadIdx.x;
	unsigned int threadCompInx = blockIdx.x * blockDim.x+ threadIdx.x;
	//All modern GPU have warp sizeOfInt 32
	int warpIndex = threadCompInx / warpSize;

	int threadIndegOfWarp = threadCompInx & (warpSize - 1);
	//unsigned int gridSize = blockSize * gridDim.x * 2;

	data[0] = 0;
	int index = warpIndex;

	if (index < M)
	{

		data[threadIdx.x] = 0;
		for (unsigned j = IRP[index] + threadIdx.x; index < IRP[index + 1]; j+= warpSize)
		{
			data[threadIdx.x] += AS[j] * IN[JA[j]];
		
		}
		//__syncthreads();
		
		/*
		Experimental solution not finished

		//Block sizeOfInt up to 512
		if (blockSize >= 512)
		{
			if (threadIndex < 256)
			{
				data[threadIndex] += data[threadIndex + 256];
			}
			__syncthreads();
		}

		//Block sizeOfInt up to 256
		if (blockSize >= 256)
		{
			if (threadIndex < 128)
			{
				data[threadIndex] += data[threadIndex + 128];
			}
			__syncthreads();
		}
		//Block sizeOfInt up to 128
		if (blockSize >= 128)
		{
			if (threadIndex < 64)
			{
				data[threadIndex] += data[threadIndex + 64];
			}
			__syncthreads();
		}
		*/
		


		if (threadIndex < warpSize)
		{
			
			if (threadIndegOfWarp < 16) data[threadIndex] += data[threadIndex + 16];
			if (threadIndegOfWarp < 8) data[threadIndex] += data[threadIndex + 8];
			if (threadIndegOfWarp < 4) data[threadIndex] += data[threadIndex + 4];
			if (threadIndegOfWarp < 2) data[threadIndex] += data[threadIndex + 2];
			if (threadIndegOfWarp < 1) data[threadIndex] += data[threadIndex + 1];
		}

		//First thread stores output
		if (threadIndex == 0)
		{
			OUT[index] = data[threadIdx.x];
		}

	}

	
	
}


__global__ void ELLPackCudaMatrixVectorProduct(const int &M, const int & NZ, const int * JA, const double * AS, double * IN, double * OUT, int & maxBlocks)
{
	//Based on local memory
	extern __shared__ double data[];

	unsigned int block = blockIdx.x;
	unsigned int warpSize = 32;

	while (block < NZ)
	{
		unsigned int threadIndex = threadIdx.x;
		unsigned int threadCompIdx = blockIdx.x * NZ + threadIdx.x;
		//All modern GPU have warp sizeOfInt 32
		unsigned int warpIndex = threadCompIdx / warpSize;
		unsigned int threadIndegOfWarp = threadCompIdx & (warpSize - 1);
		//unsigned int gridSize = blockSize * gridDim.x * 2;

		unsigned int limit = blockDim.x / 2;
		data[threadIdx.x] = 0;

		while (threadIndex < NZ)
		{
			data[threadIdx.x] += AS[threadCompIdx] * IN[JA[threadCompIdx]];

			threadIndex += blockDim.x;
			threadCompIdx += blockDim.x;
		}
		__syncthreads();

		// data, OUT[block], limit
		while (limit > 0)
		{
			if (threadIdx.x < limit)
			{
				data[threadIdx.x] = data[threadIdx.x] + data[threadIdx.x + limit];
			}

			__syncthreads();

			limit = limit / 2;
		}

		if (threadIdx.x == 0)
		{
			OUT[block] = data[0];
		}


		block += maxBlocks;
		
	}
	


}





void CUDASolver(ReadMatrixCSR &mat, std::vector<double> &  X, std::vector<double>& Y, int blockSize, int maxBlocks, double & timeToComplete)
{

	//Size variables
	int M = mat.getM();
	int NZ = mat.getNZ();
	int N = mat.getN();

	//Host variables
	auto hostJA = mat.getJA();
	auto hostIRP = mat.getIRP();
	auto hostAS = mat.getAS();
	//Variables for host array
	int hostArrayM[5];
	hostArrayM[0] = M;
	Y.resize(NZ);
	std::fill(Y.begin(), Y.end(), 0);

	//device variables
	int * d_runParam = 0;
	int * d_JA = 0;
	int * d_IRP = 0;
	double * d_AS = 0;
	double * d_X = 0;
	double * d_Y = 0;


	hipError_t cudaStatus;
	//One dimensional M number of blocks, one block one Row 
	dim3 numberOfBlocks(M, 1, 1);
	dim3 thredsInBlock(blockSize, 1, 1);

	
	int sizeOfInt = sizeof(int);
	int sizeDouble = sizeof(double);
	
	//Time measurements variables
	hipEvent_t start;
	hipEvent_t end;
	float totalTimeOfExtecution = 8;
	
	hipEventCreate(&start);
	hipEventCreate(&end);


	try
	{

		cudaStatus = hipSetDevice(0);



		//Allocation of arrays to GPU , Host to GPU
		cudaStatus = hipMalloc((void**)&d_runParam, sizeOfInt * 2);
		cudaStatus = hipMalloc((void**)&d_JA, sizeOfInt * NZ);
		cudaStatus = hipMalloc((void**)&d_IRP, sizeOfInt * (M + 1));
		cudaStatus = hipMalloc((void**)&d_AS, sizeDouble* NZ);
		cudaStatus = hipMalloc((void**)&d_Y, sizeDouble * N);
		cudaStatus = hipMalloc((void**)&d_X, sizeDouble * N);



		//Copy data from host to device memory
		cudaStatus = hipMemcpy(d_runParam, &hostArrayM[0], sizeOfInt * 2, hipMemcpyHostToDevice);
		cudaStatus = hipMemcpy(d_JA, &hostJA, sizeOfInt * NZ, hipMemcpyHostToDevice);
		cudaStatus = hipMemcpy(d_IRP, &hostIRP, sizeOfInt * (M + 1), hipMemcpyHostToDevice);
		cudaStatus = hipMemcpy(d_AS, &hostAS, sizeDouble* NZ, hipMemcpyHostToDevice);
		cudaStatus = hipMemcpy(d_X, &X[0], sizeDouble * N, hipMemcpyHostToDevice);


		//Start time Measurement 
		hipEventRecord(start);
		//Run kernel (const int &M, const int * JA, const int * IRP, const int * AS, double * OUT, double * IN)
		CSRCudaMatrixVectorProduct << <numberOfBlocks, thredsInBlock, blockSize * sizeDouble >> > (M, d_JA, d_IRP, d_AS, d_X, d_Y);

		cudaStatus = hipGetLastError();
		if (cudaStatus != hipSuccess)
		{
			throw std::exception("FAILED TO RUN KERNEL");
			std::cout << "KERNEL RUN FAILED" << std::endl;
		}

		hipEventRecord(end);

		cudaStatus = hipGetLastError();
		cudaStatus = hipDeviceSynchronize();

		//copy resulst to host
		cudaStatus = hipMemcpy(&Y[0], d_Y, N * sizeDouble, hipMemcpyDeviceToHost);


		hipEventSynchronize(end);
		hipEventElapsedTime(&totalTimeOfExtecution, start, end);
	

	}


	catch (const std::exception & ex)
	{
		hipFree(d_runParam);
		hipFree(d_JA);
		hipFree(d_IRP);
		hipFree(d_AS);
		hipFree(d_X);
		hipFree(d_Y);
	}
	
}



	



void CUDASolver(ReadMatrixELL & mat, std::vector<double>& X, std::vector<double>& Y, int blockSize, int maxBlocks, double & timeToComplete)
{

	//Size variables
	int M = mat.getM();
	int NZ = mat.getNZ();
	int N = mat.getN();

	//Host varables
	auto hostJA = mat.getJA();
	auto hostAS = mat.getAS();
	//Varaibles for host arrary
	int hostArrayM[3];
	hostArrayM[0] = M;
	Y.resize(NZ);
	std::fill(Y.begin(), Y.end(), 0);

	//device variables
	int * d_runParam = 0;
	int * d_JA = 0;
	double * d_AS = 0;
	double * d_X = 0;
	double * d_Y = 0;


	hipError_t cudaStatus;
	//One dimensional M number of blocks, one block one Row 
	dim3 numberOfBlocks(M, 1, 1);
	dim3 thredsInBlock(blockSize, 1, 1);


	int sizeOfInt = sizeof(int);
	int sizeDouble = sizeof(double);

	//Time measurements variables
	hipEvent_t start;
	hipEvent_t end;
	float totalTimeOfExtecution = 8.0;

	hipEventCreate(&start);
	hipEventCreate(&end);


	try
	{

		cudaStatus = hipSetDevice(0);



		//Allocation of arrays to GPU , Host to GPU
		cudaStatus = hipMalloc((void**)&d_runParam, sizeOfInt * 3);
		cudaStatus = hipMalloc((void**)&d_JA, sizeOfInt * NZ);
		cudaStatus = hipMalloc((void**)&d_AS, sizeDouble* NZ);
		cudaStatus = hipMalloc((void**)&d_Y, sizeDouble * N);
		cudaStatus = hipMalloc((void**)&d_X, sizeDouble * N);



		//Copy data from host to device memory
		cudaStatus = hipMemcpy(d_runParam, &hostArrayM[0], sizeOfInt * 3, hipMemcpyHostToDevice);
		cudaStatus = hipMemcpy(d_JA, &hostJA, sizeOfInt * NZ, hipMemcpyHostToDevice);
		cudaStatus = hipMemcpy(d_AS, &hostAS, sizeDouble* NZ, hipMemcpyHostToDevice);
		cudaStatus = hipMemcpy(d_X, &X[0], sizeDouble * N, hipMemcpyHostToDevice);


		//Start time Measurement 
		hipEventRecord(start);
		//Run kernel (const int &M, const int * JA, const int * IRP, const int * AS, double * OUT, double * IN)
		ELLPackCudaMatrixVectorProduct<< <numberOfBlocks, thredsInBlock, blockSize * sizeDouble >> > (M, NZ, d_JA, d_AS, d_X, d_Y, maxBlocks);

		hipEventRecord(end);

		cudaStatus = hipGetLastError();
		cudaStatus = hipDeviceSynchronize();

		//copy resulst to host
		cudaStatus = hipMemcpy(&Y[0], d_Y, N * sizeDouble, hipMemcpyDeviceToHost);


		//hipEventSynchronize(end);
		//hipEventElapsedTime(&totalTimeOfExtecution, start, end);


	}


	catch (const std::exception & ex)
	{
		hipFree(d_runParam);
		hipFree(d_JA);
		hipFree(d_AS);
		hipFree(d_X);
		hipFree(d_Y);
	}

}

