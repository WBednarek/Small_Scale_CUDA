#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "ReadMatrix.h"


/**
*
* @param M Numner of rows
* @param JA Array of column indieces
* @param IRP Array of pointers to row start
* @param AS Array of cooefficients
* @param OUT Solution vector
* @param blockAmount
*/


__global__ void CSRCudaMatrixVectorProduct(const int &M, const int * JA, const int * IRP, const double * AS, double * IN, double * OUT)
{

	extern __shared__ double data[];

	
	unsigned int threadIndex = threadIdx.x;
	unsigned int t_idx = blockIdx.x * blockDim.x+ threadIndex;
	//All modern GPU have warp size 32
	int w_idx = t_idx / 32;

	int tw_idx = t_idx & (32 - 1);
	//unsigned int gridSize = blockSize * gridDim.x * 2;

	data[0] = 0;
	int i = w_idx;

	if (i < M)
	{

		data[threadIndex] = 0;
		for (unsigned j = IRP[i] + threadIdx.x; i < IRP[i + 1]; j+=32)
		{
			data[threadIndex] += AS[j] * IN[JA[j]];
		
		}
		__syncthreads();
		/*
		//Block size up to 512
		if (blockSize >= 512)
		{
			if (threadIndex < 256)
			{
				data[threadIndex] += data[threadIndex + 256];
			}
			__syncthreads();
		}

		//Block size up to 256
		if (blockSize >= 256)
		{
			if (threadIndex < 128)
			{
				data[threadIndex] += data[threadIndex + 128];
			}
			__syncthreads();
		}
		//Block size up to 128
		if (blockSize >= 128)
		{
			if (threadIndex < 64)
			{
				data[threadIndex] += data[threadIndex + 64];
			}
			__syncthreads();
		}
		*/
		


		if (threadIndex < 32)
		{
			
			if (tw_idx < 16) data[threadIndex] += data[threadIndex + 16];
			if (tw_idx < 8) data[threadIndex] += data[threadIndex + 8];
			if (tw_idx < 4) data[threadIndex] += data[threadIndex + 4];
			if (tw_idx < 2) data[threadIndex] += data[threadIndex + 2];
			if (tw_idx < 1) data[threadIndex] += data[threadIndex + 1];
		}

		//First thread stores output
		if (threadIndex == 0)
		{
			OUT[i] = data[threadIdx.x];
		}

	}

	
	
}



void CRSCUDASolver(ReadMatrixCSR &mat,  std::vector<double> & X, std::vector<double> & Y, float exeTime, int blockSize, int maxBlocks )
{

	//Host varables
	auto hostJA = mat.getJA();
	auto hostIRP = mat.getIRP();
	auto hostAS = mat.getAS();

	//device variables
	int * d_runParam = 0;
	int * d_JA;
	int * d_IRP;
	double * d_AS;
	double * d_X;
	double * d_Y;

	//Size variables
	int M = mat.getM();
	int NZ = mat.getNZ();
	int N = mat.getN();
	hipError_t cudaStatus;
	//One dimensional M number of blocks, one block one Row 
	dim3 numberOfBlocks(M, 1, 1);
	dim3 thredsInBlock(blockSize, 1, 1);
	
	int size = sizeof(int);
	int sizeDouble = sizeof(double);
	cudaStatus = hipSetDevice(0);

	//Varaibles for host arrary
	int hostArrayM[5];
	hostArrayM[0] = M;

	//Allocation of arrays to GPU , Host to GPU
	cudaStatus = hipMalloc((void**)&d_runParam, size*2);
	cudaStatus = hipMalloc((void**)&d_JA, size * NZ);
	cudaStatus = hipMalloc((void**)&d_IRP, size * (M+1));
	cudaStatus = hipMalloc((void**)&d_AS, sizeDouble* NZ);
	cudaStatus = hipMalloc((void**)&d_Y, sizeDouble * N);
	cudaStatus = hipMalloc((void**)&d_X, sizeDouble * N);



	//Copy data from host to device memory
	cudaStatus = hipMemcpy(d_runParam, &hostArrayM[0], size * 2, hipMemcpyHostToDevice);
	cudaStatus = hipMemcpy(d_JA, &hostJA, size * NZ, hipMemcpyHostToDevice);
	cudaStatus = hipMemcpy(d_IRP, &hostIRP, size * (M + 1), hipMemcpyHostToDevice);
	cudaStatus = hipMemcpy(d_AS, &hostAS, sizeDouble* NZ, hipMemcpyHostToDevice);
	cudaStatus = hipMemcpy(d_X, &X[0], sizeDouble * N, hipMemcpyHostToDevice);

	//Run kernel (const int &M, const int * JA, const int * IRP, const int * AS, double * OUT, double * IN)
	CSRCudaMatrixVectorProduct << <numberOfBlocks, thredsInBlock, blockSize * sizeDouble >> > (M, d_JA, d_IRP, d_AS, d_X, d_Y);


	//copy resulst to host
	cudaStatus = hipMemcpy(&Y[0], d_Y, N * sizeDouble, hipMemcpyDeviceToHost);
}
	


