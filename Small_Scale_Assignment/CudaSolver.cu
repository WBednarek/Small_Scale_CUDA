#include "hip/hip_runtime.h"
#include "CudaSolver.h"


/**
*
* @param M Number of rows
* @param JA Array of column JA
* @param IRP Array of pointers to selected start
* @param AS Array of coefficients
* @param OUT Solution vector
* @param blockAmount
*/


__global__ void CSRCudaMatrixVectorProduct(const int &M, const int * JA, const int * IRP, const double * AS, double * IN, double * OUT)
{

	//Based on local memory
	extern __shared__ double storeArray[];

	unsigned int warpSize = 32; //All modern GPU have warp sizeOfInt 32
	unsigned int threadID = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned int warpID = threadID / warpSize;
	unsigned int threadIndOfWarp = threadID & (warpSize - 1);

	unsigned int threadIndex = threadIdx.x;
	
	unsigned int selected = warpID;
		

	//int warpIndex = threadID / warpSize;

	//int threadIndegOfWarp = threadID & (warpSize - 1);
	//unsigned int gridSize = blockSize * gridDim.x * 2;

	//storeArray[0] = 0;
	//int index = warpIndex;

	if (selected < M)
	{
		unsigned int rowBegining = IRP[selected];
		unsigned int endOfRow = IRP[selected + 1];

		storeArray[threadIdx.x] = 0;
		for (unsigned j = rowBegining + threadIndOfWarp; j < endOfRow; j+= warpSize)
		{
			storeArray[threadIdx.x] += AS[j] * IN[JA[j]];
		
		}


		if (threadIndOfWarp < 16) storeArray[threadIdx.x] += storeArray[threadIdx.x + 16];
		if (threadIndOfWarp < 8) storeArray[threadIdx.x] += storeArray[threadIdx.x + 8];
		if (threadIndOfWarp < 4) storeArray[threadIdx.x] += storeArray[threadIdx.x + 4];
		if (threadIndOfWarp < 2) storeArray[threadIdx.x] += storeArray[threadIdx.x + 2];
		if (threadIndOfWarp < 1) storeArray[threadIdx.x] += storeArray[threadIdx.x + 1];


		if (threadIndOfWarp == 0)
		{
			OUT[selected] = storeArray[threadIdx.x];
		}

	
		//First thread stores output
		

		//__syncthreads();
		
		/*
		Experimental solution not finished

		//Block sizeOfInt up to 512
		if (blockSize >= 512)
		{
			if (threadIndex < 256)
			{
				storeArray[threadIndex] += storeArray[threadIndex + 256];
			}
			__syncthreads();
		}

		//Block sizeOfInt up to 256
		if (blockSize >= 256)
		{
			if (threadIndex < 128)
			{
				storeArray[threadIndex] += storeArray[threadIndex + 128];
			}
			__syncthreads();
		}
		//Block sizeOfInt up to 128
		if (blockSize >= 128)
		{
			if (threadIndex < 64)
			{
				storeArray[threadIndex] += storeArray[threadIndex + 64];
			}
			__syncthreads();
		}
		*/
		


		

	}

	
	
}


__global__ void ELLPackCudaMatrixVectorProduct(const int &M, const int & NZ, const int * JA, const double * AS, double * IN, double * OUT, int & maxBlocks)
{
	//Based on local memory
	extern __shared__ double storeArray[];

	unsigned int block = blockIdx.x;
	unsigned int warpSize = 32;

	while (block < NZ)
	{
		unsigned int threadIndex = threadIdx.x;
		unsigned int threadCompIdx = blockIdx.x * NZ + threadIdx.x;
		//All modern GPU have warp sizeOfInt 32
		unsigned int warpIndex = threadCompIdx / warpSize;
		unsigned int threadIndegOfWarp = threadCompIdx & (warpSize - 1);
		//unsigned int gridSize = blockSize * gridDim.x * 2;

		unsigned int limit = blockDim.x / 2;
		storeArray[threadIdx.x] = 0;

		while (threadIndex < NZ)
		{
			storeArray[threadIdx.x] += AS[threadCompIdx] * IN[JA[threadCompIdx]];

			threadIndex += blockDim.x;
			threadCompIdx += blockDim.x;
		}
		__syncthreads();

		// storeArray, OUT[block], limit
		while (limit > 0)
		{
			if (threadIdx.x < limit)
			{
				storeArray[threadIdx.x] = storeArray[threadIdx.x] + storeArray[threadIdx.x + limit];
			}

			__syncthreads();

			limit = limit / 2;
		}

		if (threadIdx.x == 0)
		{
			OUT[block] = storeArray[0];
		}


		block += maxBlocks;
		
	}
	


}





void CUDASolver(ReadMatrixCSR &mat, std::vector<double> &  X, std::vector<double>& Y, int blockSize, int maxBlocks, double & timeToComplete)
{

	//Size variables
	int M = mat.getM();
	int NZ = mat.getNZ();
	int N = mat.getN();

	//Host variables
	auto hostJA = mat.getJA();
	auto hostIRP = mat.getIRP();
	auto hostAS = mat.getAS();
	//Variables for host array
	int hostArrayM[5];
	hostArrayM[0] = M;
	Y.resize(NZ);
	std::fill(Y.begin(), Y.end(), 0);

	//device variables
	int * d_runParam = 0;
	int * d_JA = 0;
	int * d_IRP = 0;
	double * d_AS = 0;
	double * d_X = 0;
	double * d_Y = 0;


	hipError_t cudaStatus;
	//One dimensional M number of blocks, one block one Row 
	dim3 numberOfBlocks(M, 1, 1);
	dim3 thredsInBlock(blockSize, 1, 1);

	
	int sizeOfInt = sizeof(int);
	int sizeDouble = sizeof(double);
	
	//Time measurements variables
	hipEvent_t start;
	hipEvent_t end;
	float totalTimeOfExtecution = 8;
	
	hipEventCreate(&start);
	hipEventCreate(&end);


	try
	{

		cudaStatus = hipSetDevice(0);



		//Allocation of arrays to GPU , Host to GPU
		cudaStatus = hipMalloc((void**)&d_runParam, sizeOfInt * 2);
		cudaStatus = hipMalloc((void**)&d_JA, sizeOfInt * NZ);
		cudaStatus = hipMalloc((void**)&d_IRP, sizeOfInt * (M + 1));
		cudaStatus = hipMalloc((void**)&d_AS, sizeDouble* NZ);
		cudaStatus = hipMalloc((void**)&d_Y, sizeDouble * N);
		cudaStatus = hipMalloc((void**)&d_X, sizeDouble * N);



		//Copy storeArray from host to device memory
		cudaStatus = hipMemcpy(d_runParam, &hostArrayM[0], sizeOfInt * 2, hipMemcpyHostToDevice);
		cudaStatus = hipMemcpy(d_JA, &hostJA, sizeOfInt * NZ, hipMemcpyHostToDevice);
		cudaStatus = hipMemcpy(d_IRP, &hostIRP, sizeOfInt * (M + 1), hipMemcpyHostToDevice);
		cudaStatus = hipMemcpy(d_AS, &hostAS, sizeDouble* NZ, hipMemcpyHostToDevice);
		cudaStatus = hipMemcpy(d_X, &X[0], sizeDouble * N, hipMemcpyHostToDevice);


		//Start time Measurement 
		hipEventRecord(start);
		//Run kernel (const int &M, const int * JA, const int * IRP, const int * AS, double * OUT, double * IN)
		CSRCudaMatrixVectorProduct << <numberOfBlocks, thredsInBlock, blockSize * sizeDouble >> > (M, d_JA, d_IRP, d_AS, d_X, d_Y);

		cudaStatus = hipGetLastError();
		if (cudaStatus != hipSuccess)
		{
			throw std::exception("FAILED TO RUN KERNEL");
			std::cout << "KERNEL RUN FAILED" << std::endl;
		}

		hipEventRecord(end);

		cudaStatus = hipGetLastError();
		cudaStatus = hipDeviceSynchronize();

		//copy resulst to host
		cudaStatus = hipMemcpy(&Y[0], d_Y, N * sizeDouble, hipMemcpyDeviceToHost);


		hipEventSynchronize(end);
		hipEventElapsedTime(&totalTimeOfExtecution, start, end);
	

	}


	catch (const std::exception & ex)
	{
		hipFree(d_runParam);
		hipFree(d_JA);
		hipFree(d_IRP);
		hipFree(d_AS);
		hipFree(d_X);
		hipFree(d_Y);
	}
	
}



	



void CUDASolver(ReadMatrixELL & mat, std::vector<double>& X, std::vector<double>& Y, int blockSize, int maxBlocks, double & timeToComplete)
{

	//Size variables
	int M = mat.getM();
	int NZ = mat.getNZ();
	int N = mat.getN();

	//Host varables
	auto hostJA = mat.getJA();
	auto hostAS = mat.getAS();
	//Varaibles for host arrary
	int hostArrayM[3];
	hostArrayM[0] = M;
	Y.resize(NZ);
	std::fill(Y.begin(), Y.end(), 0);

	//device variables
	int * d_runParam = 0;
	int * d_JA = 0;
	double * d_AS = 0;
	double * d_X = 0;
	double * d_Y = 0;


	hipError_t cudaStatus;
	//One dimensional M number of blocks, one block one Row 
	dim3 numberOfBlocks(M, 1, 1);
	dim3 thredsInBlock(blockSize, 1, 1);


	int sizeOfInt = sizeof(int);
	int sizeDouble = sizeof(double);

	//Time measurements variables
	hipEvent_t start;
	hipEvent_t end;
	float totalTimeOfExtecution = 8.0;

	hipEventCreate(&start);
	hipEventCreate(&end);


	try
	{

		cudaStatus = hipSetDevice(0);



		//Allocation of arrays to GPU , Host to GPU
		cudaStatus = hipMalloc((void**)&d_runParam, sizeOfInt * 3);
		cudaStatus = hipMalloc((void**)&d_JA, sizeOfInt * NZ);
		cudaStatus = hipMalloc((void**)&d_AS, sizeDouble* NZ);
		cudaStatus = hipMalloc((void**)&d_Y, sizeDouble * N);
		cudaStatus = hipMalloc((void**)&d_X, sizeDouble * N);



		//Copy storeArray from host to device memory
		cudaStatus = hipMemcpy(d_runParam, &hostArrayM[0], sizeOfInt * 3, hipMemcpyHostToDevice);
		cudaStatus = hipMemcpy(d_JA, &hostJA, sizeOfInt * NZ, hipMemcpyHostToDevice);
		cudaStatus = hipMemcpy(d_AS, &hostAS, sizeDouble* NZ, hipMemcpyHostToDevice);
		cudaStatus = hipMemcpy(d_X, &X[0], sizeDouble * N, hipMemcpyHostToDevice);


		//Start time Measurement 
		hipEventRecord(start);
		//Run kernel (const int &M, const int * JA, const int * IRP, const int * AS, double * OUT, double * IN)
		ELLPackCudaMatrixVectorProduct<< <numberOfBlocks, thredsInBlock, blockSize * sizeDouble >> > (M, NZ, d_JA, d_AS, d_X, d_Y, maxBlocks);

		hipEventRecord(end);

		cudaStatus = hipGetLastError();
		cudaStatus = hipDeviceSynchronize();

		//copy resulst to host
		cudaStatus = hipMemcpy(&Y[0], d_Y, N * sizeDouble, hipMemcpyDeviceToHost);


		//hipEventSynchronize(end);
		//hipEventElapsedTime(&totalTimeOfExtecution, start, end);


	}


	catch (const std::exception & ex)
	{
		hipFree(d_runParam);
		hipFree(d_JA);
		hipFree(d_AS);
		hipFree(d_X);
		hipFree(d_Y);
	}

}

